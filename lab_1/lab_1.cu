
#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <stdio.h> 
 
__global__ void SKernel(float *a, float *b, int n) { 
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    int offset = blockDim.x * gridDim.x;
    while(idx < n) {
        b[idx] = sqrt(a[idx]);
        idx += offset;
    } 
} 
 
void Printer(float *a, int n){ 
    for (int i = 0; i < n; i++){ 
        printf("%f\n", a[i]); 
    } 
} 
 
void Assigner(float *a, int n){ 
    for (int i = 0; i < n; i++){ 
        a[i] = (float)i; 
    } 
} 
 
int main() { 
    
    int n = 100; 
    int size = n * sizeof(float);
    
    float *aDev = NULL, *bDev = NULL; 
    float *a = NULL, *b = NULL; 
    
    hipMalloc((void **) &aDev, size); 
    hipMalloc((void **) &bDev, size); 
    
    a = (float *) malloc(size); 
    b = (float *) malloc(size); 
    
    Assigner(a, n); 

    hipMemcpy(aDev, a, size, hipMemcpyHostToDevice); 
    hipMemcpy(bDev, b, size, hipMemcpyHostToDevice); 
    
    SKernel<<<256, 256>>> (aDev, bDev, n); 
    
    hipMemcpy(b, bDev, size, hipMemcpyDeviceToHost); 
    
    Printer(b, n); 
    
    hipFree(aDev); 
    hipFree(bDev); 
    
    free(a); 
    free(b); 
}